// Elapsed Real Time for input-4.txt: 1.381 seconds

#include <stdio.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>

// Size of the square we're looking for.
#define SQUARE_WIDTH 6
#define SQUARE_HEIGHT 6

// Maximum width of a row.  Makes it easier to allocate the whole
// grid contiguously.
#define MAX_WIDTH 16384

// Type used for a row of the grid.  Makes it easier to declare the
// grid as a pointer.
typedef char Row[ MAX_WIDTH ];

// Size of the grid of characters.
int rows, cols;

// Grid of letters.
Row *grid;

// Kernel, run by each thread to count complete squares in parallel.
__global__ void countSquares( int rows, int cols, bool report, int *gpuResults, Row *gridCpy ) {
  // Unique index for this worker.
  int r0 = blockDim.x * blockIdx.x + threadIdx.x;
  int c = 0;

  // Make sure I actually have something to work on.
  if ( r0 + SQUARE_HEIGHT - 1 < rows ) {
    for(int col = 0; col < cols; col++) {
	  if(col + 6 <= cols) {
	    bool check = false;//check if that grid actually makes a square or not. 
	    char letters[26] = {'a','b','c','d','e','f','g','h','i','j','k','l','m','n','o','p','q','r','s','t','u','v','w','x','y','z'}; // array of alphabets. 
	    for( int i = r0; i < r0 + 6; i++) {
		  for( int j = col; j < col + 6; j++) {
			for( int k = 0; k < 26; k++) {
			  if(gridCpy[i][j] == letters[k]) {  //if cell in grid has any aplhabet, change that alphabet bucket to 0. 
			    letters[k] = '0';//if the letter is not yet, remove it. 
			  }
			}
		  } 
	    }
		
		for(int k = 0; k < 26; k++) {
		  if(letters[k] != '0') {
		    check = false;//if any letter in the array is not 0, the grid is not a square. 
			break;
		  } else {
		    check = true;//else it is true and continue to make sure all if true. 
		  }
		}
		
		if(check == true) {
	      c += 1;//if it is a perfect square grid, increment the count for that index
		  if(report) {
		    printf("%d %d\n", r0, col);//if report is true, print the index
		  }
		}
	  }
	}
  }
  gpuResults[r0] = c;//put number of squares in the array
}

// Read the grid of characters.
void readGrid() {
  // Read grid dimensions.
  scanf( "%d%d", &rows, &cols );
  if ( cols > MAX_WIDTH ) {
    fprintf( stderr, "Input grid is too wide.\n" );
    exit( EXIT_FAILURE );
  }

  // Make space to store the grid as a big, contiguous array.
  grid = (Row *) malloc( rows * sizeof( Row ) );
  
  // Read each row of the grid as a string, then copy everything
  // but the null terminator into the grid array.
  int rowCount = 0;
  char buffer[ MAX_WIDTH + 1 ];
  while ( rowCount < rows ) {
    scanf( "%s", buffer );
    memcpy( grid[ rowCount++ ], buffer, cols );
  }
}

// General function to report a failure and exit.
static void fail( char const *message ) {
  fprintf( stderr, "%s\n", message );
  exit( 1 );
}

// Print out a usage message, then exit.
static void usage() {
  printf( "usage: square [report]\n" );
  exit( 1 );
}

//main
int main( int argc, char *argv[] ) {
  // If there's an argument, it better be "report"
  bool report = false;
  if ( argc == 2 ) {
    if ( strcmp( argv[ 1 ], "report" ) != 0 )
      usage();
    report = true;
  }

  readGrid();
  
  // Need to add code to allocate memory on the device and copy the grid
  // over.
  Row *gridCpy = NULL;
  hipMalloc( (void **)&gridCpy,  rows * sizeof( Row ) );
  hipMemcpy( gridCpy, grid, rows * sizeof( Row ), hipMemcpyHostToDevice );
  
  //allocate memory for results array
  int *gpuResults = NULL;
  hipMalloc((void **)&gpuResults, rows * sizeof(int));

  // Block and grid dimensions.
  int threadsPerBlock = 250;
  // Round up.
  int blocksPerGrid = ( rows + threadsPerBlock - 1 ) / threadsPerBlock;

  // Run our kernel on these block/grid dimensions
  countSquares<<<blocksPerGrid, threadsPerBlock>>>( rows, cols, report, gpuResults, gridCpy );
  if ( hipGetLastError() != hipSuccess )
    fail( "Failure in CUDA kernel execution." );

  int *results = (int *) malloc( rows * sizeof( int ) );

  // Need to add code to copy the results list back to the host and
  // add them up.
  hipMemcpy( results, gpuResults, rows * sizeof(int), hipMemcpyDeviceToHost);
  int total = 0;
  for(int i = 0; i < rows; i++) {
    total += results[i];//add all integers in results array to get total 
  }
  printf( "Squares: %d\n", total );

  // Free memory on the device and the host.
  free(grid);
  free(results);
  hipFree(gridCpy);
  hipFree(gpuResults);

  hipDeviceReset();
  
  return 0;
}
